#include "hip/hip_runtime.h"
#include "CudaHelper.cuh"
#include "CudaInterface.hpp"

#define MAX3(a, b, c) fmaxf(fmaxf(a, b), c)
#define MIN3(a, b, c) fminf(fminf(a, b), c)

template <typename T>
__inline__ __device__ static void getLightest(
    typename Vec4<T>::type &mc, typename Vec4<T>::type &a, typename Vec4<T>::type &b, typename Vec4<T>::type &c, float strength)
{
    constexpr float offset = std::is_floating_point<T>::value ? 0.0f : 0.5f;
    mc = makeVec4<T>(
        mc.x + strength * (__fdividef(a.x + b.x + c.x, 3.0f) - mc.x) + offset,
        mc.y + strength * (__fdividef(a.y + b.y + c.y, 3.0f) - mc.y) + offset,
        mc.z + strength * (__fdividef(a.z + b.z + c.z, 3.0f) - mc.z) + offset,
        mc.w + strength * (__fdividef(a.w + b.w + c.w, 3.0f) - mc.w) + offset);
}

template <typename T>
__inline__ __device__ static void getAVerage(
    typename Vec4<T>::type &mc, typename Vec4<T>::type &a, typename Vec4<T>::type &b, typename Vec4<T>::type &c, float strength)
{
    constexpr float offset = std::is_floating_point<T>::value ? 0.0f : 0.5f;
    mc = makeVec4<T>(
        mc.x + strength * (__fdividef(a.x + b.x + c.x, 3.0f) - mc.x) + offset,
        mc.y + strength * (__fdividef(a.y + b.y + c.y, 3.0f) - mc.y) + offset,
        mc.z + strength * (__fdividef(a.z + b.z + c.z, 3.0f) - mc.z) + offset,
        0.299f * mc.z + 0.587f * mc.y + 0.114f * mc.x + offset);
}

template <typename T>
__global__ static void getGray(
    hipTextureObject_t srcImg, hipSurfaceObject_t dstImg,
    unsigned int W, unsigned int H)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= W || y >= H)
        return;

    const float u = __fdividef(x + 0.5f, W);
    const float v = __fdividef(y + 0.5f, H);
    constexpr float scale = PixelValue<T>::max();
    constexpr float offset = std::is_floating_point<T>::value ? 0.0f : 0.5f;

    float4 fmc = tex2D<float4>(srcImg, u, v);

    auto mc = makeVec4<T>(
        fmc.x * scale + offset, fmc.y * scale + offset, fmc.z * scale + offset, fmc.w * scale + offset);
    mc.w = 0.299f * mc.z + 0.587f * mc.y + 0.114f * mc.x + offset;

    surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
}

template <typename T>
__global__ static void pushColor(
    hipSurfaceObject_t srcImg, hipSurfaceObject_t dstImg,
    unsigned int W, unsigned int H, const float strength)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= W || y >= H)
        return;

    typename Vec4<T>::type tl, tc, tr, ml, mc, mr, bl, bc, br;
    surf2Dread(&tl, srcImg, sizeof(mc) * (x - 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&tc, srcImg, sizeof(mc) * x, y - 1, hipBoundaryModeZero);
    surf2Dread(&tr, srcImg, sizeof(mc) * (x + 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&ml, srcImg, sizeof(mc) * (x - 1), y, hipBoundaryModeZero);
    surf2Dread(&mc, srcImg, sizeof(mc) * x, y, hipBoundaryModeZero);
    surf2Dread(&mr, srcImg, sizeof(mc) * (x + 1), y, hipBoundaryModeZero);
    surf2Dread(&bl, srcImg, sizeof(mc) * (x - 1), y + 1, hipBoundaryModeZero);
    surf2Dread(&bc, srcImg, sizeof(mc) * x, y + 1, hipBoundaryModeZero);
    surf2Dread(&br, srcImg, sizeof(mc) * (x + 1), y + 1, hipBoundaryModeZero);

    T maxD, minL;

    //top and bottom
    maxD = MAX3(bl.w, bc.w, br.w);
    minL = MIN3(tl.w, tc.w, tr.w);
    if (minL > mc.w && mc.w > maxD)
        getLightest<T>(mc, tl, tc, tr, strength);
    else
    {
        maxD = MAX3(tl.w, tc.w, tr.w);
        minL = MIN3(bl.w, bc.w, br.w);
        if (minL > mc.w && mc.w > maxD)
            getLightest<T>(mc, bl, bc, br, strength);
    }

    //sundiagonal
    maxD = MAX3(ml.w, mc.w, bc.w);
    minL = MIN3(tc.w, tr.w, mr.w);
    if (minL > maxD)
        getLightest<T>(mc, tc, tr, mr, strength);
    else
    {
        maxD = MAX3(tc.w, mc.w, mr.w);
        minL = MIN3(ml.w, bl.w, bc.w);
        if (minL > maxD)
            getLightest<T>(mc, ml, bl, bc, strength);
    }

    //left and right
    maxD = MAX3(tl.w, ml.w, bl.w);
    minL = MIN3(tr.w, mr.w, br.w);
    if (minL > mc.w && mc.w > maxD)
        getLightest<T>(mc, tr, mr, br, strength);
    else
    {
        maxD = MAX3(tr.w, mr.w, br.w);
        minL = MIN3(tl.w, ml.w, bl.w);
        if (minL > mc.w && mc.w > maxD)
            getLightest<T>(mc, tl, ml, bl, strength);
    }

    //diagonal
    maxD = MAX3(tc.w, mc.w, ml.w);
    minL = MIN3(mr.w, br.w, bc.w);
    if (minL > maxD)
        getLightest<T>(mc, mr, br, bc, strength);
    else
    {
        maxD = MAX3(bc.w, mc.w, mr.w);
        minL = MIN3(ml.w, tl.w, tc.w);
        if (minL > maxD)
            getLightest<T>(mc, ml, tl, tc, strength);
    }

    surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
}

template <typename T>
__global__ static void getGradient(
    hipSurfaceObject_t srcImg, hipSurfaceObject_t dstImg,
    unsigned int W, unsigned int H)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= W || y >= H)
        return;

    typename Vec4<T>::type tl, tc, tr, ml, mc, mr, bl, bc, br;
    surf2Dread(&tl, srcImg, sizeof(mc) * (x - 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&tc, srcImg, sizeof(mc) * x, y - 1, hipBoundaryModeZero);
    surf2Dread(&tr, srcImg, sizeof(mc) * (x + 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&ml, srcImg, sizeof(mc) * (x - 1), y, hipBoundaryModeZero);
    surf2Dread(&mc, srcImg, sizeof(mc) * x, y, hipBoundaryModeZero);
    surf2Dread(&mr, srcImg, sizeof(mc) * (x + 1), y, hipBoundaryModeZero);
    surf2Dread(&bl, srcImg, sizeof(mc) * (x - 1), y + 1, hipBoundaryModeZero);
    surf2Dread(&bc, srcImg, sizeof(mc) * x, y + 1, hipBoundaryModeZero);
    surf2Dread(&br, srcImg, sizeof(mc) * (x + 1), y + 1, hipBoundaryModeZero);

    const float gradX = tr.w + mr.w + mr.w + br.w - tl.w - ml.w - ml.w - bl.w;
    const float gradY = tl.w + tc.w + tc.w + tr.w - bl.w - bc.w - bc.w - br.w;

    mc.w = PixelValue<T>::max() - clamp(sqrtf(gradX * gradX + gradY * gradY), PixelValue<T>::min(), PixelValue<T>::max());

    surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
}

template <typename T>
__global__ static void pushGradient(
    hipSurfaceObject_t srcImg, hipSurfaceObject_t dstImg,
    unsigned int W, unsigned int H, const float strength)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= W || y >= H)
        return;

    typename Vec4<T>::type tl, tc, tr, ml, mc, mr, bl, bc, br;
    surf2Dread(&tl, srcImg, sizeof(mc) * (x - 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&tc, srcImg, sizeof(mc) * x, y - 1, hipBoundaryModeZero);
    surf2Dread(&tr, srcImg, sizeof(mc) * (x + 1), y - 1, hipBoundaryModeZero);
    surf2Dread(&ml, srcImg, sizeof(mc) * (x - 1), y, hipBoundaryModeZero);
    surf2Dread(&mc, srcImg, sizeof(mc) * x, y, hipBoundaryModeZero);
    surf2Dread(&mr, srcImg, sizeof(mc) * (x + 1), y, hipBoundaryModeZero);
    surf2Dread(&bl, srcImg, sizeof(mc) * (x - 1), y + 1, hipBoundaryModeZero);
    surf2Dread(&bc, srcImg, sizeof(mc) * x, y + 1, hipBoundaryModeZero);
    surf2Dread(&br, srcImg, sizeof(mc) * (x + 1), y + 1, hipBoundaryModeZero);

    T maxD, minL;

    //top and bottom
    maxD = MAX3(bl.w, bc.w, br.w);
    minL = MIN3(tl.w, tc.w, tr.w);
    if (minL > mc.w && mc.w > maxD)
    {
        getAVerage<T>(mc, tl, tc, tr, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    maxD = MAX3(tl.w, tc.w, tr.w);
    minL = MIN3(bl.w, bc.w, br.w);
    if (minL > mc.w && mc.w > maxD)
    {
        getAVerage<T>(mc, bl, bc, br, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    //sundiagonal
    maxD = MAX3(ml.w, mc.w, bc.w);
    minL = MIN3(tc.w, tr.w, mr.w);
    if (minL > maxD)
    {
        getAVerage<T>(mc, tc, tr, mr, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    maxD = MAX3(tc.w, mc.w, mr.w);
    minL = MIN3(ml.w, bl.w, bc.w);
    if (minL > maxD)
    {
        getAVerage<T>(mc, ml, bl, bc, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    //left and right
    maxD = MAX3(tl.w, ml.w, bl.w);
    minL = MIN3(tr.w, mr.w, br.w);
    if (minL > mc.w && mc.w > maxD)
    {
        getAVerage<T>(mc, tr, mr, br, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    maxD = MAX3(tr.w, mr.w, br.w);
    minL = MIN3(tl.w, ml.w, bl.w);
    if (minL > mc.w && mc.w > maxD)
    {
        getAVerage<T>(mc, tl, ml, bl, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    //diagonal
    maxD = MAX3(tc.w, mc.w, ml.w);
    minL = MIN3(mr.w, br.w, bc.w);
    if (minL > maxD)
    {
        getAVerage<T>(mc, mr, br, bc, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }
    maxD = MAX3(bc.w, mc.w, mr.w);
    minL = MIN3(ml.w, tl.w, tc.w);
    if (minL > maxD)
    {
        getAVerage<T>(mc, ml, tl, tc, strength);
        surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
        return;
    }

    mc.w = 0.299f * mc.z + 0.587f * mc.y + 0.114f * mc.x;
    surf2Dwrite(mc, dstImg, sizeof(mc) * x, y, hipBoundaryModeZero);
}

template <typename T>
static void cuRunKernelAnime4K09Impl(const T *inputData, T *outputData, Anime4KCPP::Cuda::ACCudaParamAnime4K09 *param)
{
    hipError_t err = hipSuccess;
    if (currCudaDeviceID)
    {
        err = hipSetDevice(currCudaDeviceID);
        CheckCudaErr(err);
    }

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<typename Vec4<T>::type>();

    hipArray_t cuArray0;
    err = hipMallocArray(&cuArray0, &channelDesc, param->orgW, param->orgH);
    CheckCudaErr(err);

    hipArray_t cuArray1;
    err = hipMallocArray(&cuArray1, &channelDesc, param->W, param->H, hipArraySurfaceLoadStore);
    CheckCudaErr(err);

    hipArray_t cuArray2;
    err = hipMallocArray(&cuArray2, &channelDesc, param->W, param->H, hipArraySurfaceLoadStore);
    CheckCudaErr(err);

    hipArray_t cuArray3;
    err = hipMallocArray(&cuArray3, &channelDesc, param->W, param->H, hipArraySurfaceLoadStore);
    CheckCudaErr(err);

    struct hipResourceDesc resDesc;
    struct hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    memset(&texDesc, 0, sizeof(texDesc));

    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = std::is_floating_point<T>::value ? hipReadModeElementType : hipReadModeNormalizedFloat;
    texDesc.normalizedCoords = 1;

    resDesc.resType = hipResourceTypeArray;

    resDesc.res.array.array = cuArray0;
    hipTextureObject_t tex = 0;
    err = hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
    CheckCudaErr(err);

    resDesc.res.array.array = cuArray1;
    hipSurfaceObject_t surf1 = 0;
    err = hipCreateSurfaceObject(&surf1, &resDesc);
    CheckCudaErr(err);

    resDesc.res.array.array = cuArray2;
    hipSurfaceObject_t surf2 = 0;
    err = hipCreateSurfaceObject(&surf2, &resDesc);
    CheckCudaErr(err);

    resDesc.res.array.array = cuArray3;
    hipSurfaceObject_t surf3 = 0;
    err = hipCreateSurfaceObject(&surf3, &resDesc);
    CheckCudaErr(err);

    err = hipMemcpy2DToArrayAsync(cuArray0, 0, 0, inputData,
                                   param->stride, sizeof(typename Vec4<T>::type) * param->orgW, param->orgH,
                                   hipMemcpyHostToDevice, stream);
    CheckCudaErr(err);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(
        (param->W + dimBlock.x - 1) / dimBlock.x,
        (param->H + dimBlock.y - 1) / dimBlock.y);

    {
        int i;
        getGray<T><<<dimGrid, dimBlock, 0, stream>>>(tex, surf1, param->W, param->H);
        for (i = 0; i < param->passes && i < param->pushColorCount; i++)
        {
            pushColor<T><<<dimGrid, dimBlock, 0, stream>>>(surf1, surf2, param->W, param->H, param->strengthColor);
            getGradient<T><<<dimGrid, dimBlock, 0, stream>>>(surf2, surf3, param->W, param->H);
            pushGradient<T><<<dimGrid, dimBlock, 0, stream>>>(surf3, surf1, param->W, param->H, param->strengthGradient);
        }
        while (i++ < param->passes)
        {
            getGradient<T><<<dimGrid, dimBlock, 0, stream>>>(surf1, surf2, param->W, param->H);
            pushGradient<T><<<dimGrid, dimBlock, 0, stream>>>(surf2, surf1, param->W, param->H, param->strengthGradient);
        }
    }

    err = hipHostRegister(outputData, sizeof(typename Vec4<T>::type) * param->W * param->H, hipHostRegisterDefault);
    CheckCudaErr(err);

    err = hipMemcpy2DFromArrayAsync(outputData, sizeof(typename Vec4<T>::type) * param->W, cuArray1, 0, 0,
                                     sizeof(typename Vec4<T>::type) * param->W, param->H,
                                     hipMemcpyDeviceToHost, stream);
    CheckCudaErr(err);

    err = hipStreamSynchronize(stream);
    CheckCudaErr(err);

    err = hipHostUnregister(outputData);
    CheckCudaErr(err);

    err = hipGetLastError();
    CheckCudaErr(err);

    hipDestroyTextureObject(tex);
    hipDestroySurfaceObject(surf1);
    hipDestroySurfaceObject(surf2);
    hipDestroySurfaceObject(surf3);

    hipFreeArray(cuArray0);
    hipFreeArray(cuArray1);
    hipFreeArray(cuArray2);
    hipFreeArray(cuArray3);

    hipStreamDestroy(stream);
}

void Anime4KCPP::Cuda::cuRunKernelAnime4K09(const void* inputData, void* outputData, ACCudaDataType type, ACCudaParamAnime4K09* param)
{
    switch (type)
    {
    case ACCudaDataType::AC_8U:
        cuRunKernelAnime4K09Impl<uchar>(reinterpret_cast<const uchar *>(inputData), reinterpret_cast<uchar *>(outputData), param);
        break;
    case ACCudaDataType::AC_16U:
        cuRunKernelAnime4K09Impl<ushort>(reinterpret_cast<const ushort *>(inputData), reinterpret_cast<ushort *>(outputData), param);
        break;
    case ACCudaDataType::AC_32F:
        cuRunKernelAnime4K09Impl<float>(reinterpret_cast<const float *>(inputData), reinterpret_cast<float *>(outputData), param);
        break;
    }
}
